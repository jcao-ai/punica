#include "hip/hip_runtime.h"
#include "sgmv_cutlass.cuh"

template bool sgmv<half>(nv_half *y, nv_half *x, nv_half **w, int32_t *s,
                            void *tmp_d, int num_problems, int d_in, int d_out,
                            int layer_idx, hipStream_t stream);

template bool sgmv<float>(float *y, float *x, float **w, int32_t *s,
                            void *tmp_d, int num_problems, int d_in, int d_out,
                            int layer_idx, hipStream_t stream);

#ifdef ENABLE_BF16
template bool sgmv<hip_bfloat16>(hip_bfloat16 *y, hip_bfloat16 *x, hip_bfloat16 **w,
                                int32_t *s, void *tmp_d, int num_problems,
                                int d_in, int d_out, int layer_idx, hipStream_t stream);
#endif